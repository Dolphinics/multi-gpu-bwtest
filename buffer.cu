#include <hip/hip_runtime.h>
#include <memory>
#include <exception>
#include <stdexcept>
#include "buffer.h"


static void deleteHostBuffer(void* buffer)
{
    hipHostFree(buffer);
}


static void deleteDeviceBuffer(void* buffer)
{
    hipFree(buffer);
}


BufferPtr createHostBuffer(size_t length, unsigned int flags)
{
    void* buffer;

    hipError_t err = hipHostAlloc(&buffer, length, flags);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return BufferPtr(buffer, &deleteHostBuffer);
}


BufferPtr createDeviceBuffer(int device, size_t length)
{
    hipError_t err;
    void* buffer;

    err = hipSetDevice(device);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    err = hipMalloc(&buffer, length);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return BufferPtr(buffer, &deleteDeviceBuffer);
}
