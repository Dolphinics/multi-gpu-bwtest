#include <hip/hip_runtime.h>
#include <vector>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include <cstdio>
#include "bench.h"
#include "buffer.h"
#include "stream.h"
#include "timer.h"

using std::vector;
using std::runtime_error;
using std::string;


static string bytesToUnit(size_t size)
{
    char buffer[1024];
    const char* units[] = { "B  ", "KiB", "MiB", "GiB", "TiB" };
    size_t i = 0, n = sizeof(units) / sizeof(units[0]);

    double csize = (double) size;

    while (i < (n - 1) && csize >= 1024.0)
    {
        csize /= 1024.0;
        ++i;
    }

    snprintf(buffer, sizeof(buffer), "%.2f %s", csize, units[i]);
    return string(buffer);
}


static string transferDirectionToString(hipMemcpyKind direction)
{
    if (direction == hipMemcpyHostToDevice)
    {
        return string("HtoD");
    }
    if (direction == hipMemcpyDeviceToHost)
    {
        return string("DtoH");
    }

    return string("unknown");
}


static void timeTransfers(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        hipStream_t stream = *spec.stream;

        const void* src = spec.direction == hipMemcpyDeviceToHost ? spec.deviceBuffer.get() : spec.hostBuffer.get();
        void* dst = spec.direction == hipMemcpyDeviceToHost ? spec.hostBuffer.get() : spec.deviceBuffer.get();

        err = hipEventRecord(spec.timer->started, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipMemcpyAsync(dst, src, spec.length, spec.direction, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventRecord(spec.timer->stopped, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


static void syncStreams(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        err = hipStreamSynchronize(*spec.stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


void runBandwidthTest(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    // Create timing events on the null stream
    TimerPtr globalTimer = createTimer();
    err = hipEventRecord(globalTimer->started);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    // Execute transfers
    try
    {
        fprintf(stdout, "Executing transfers..........");
        fflush(stdout);
        timeTransfers(transferSpecs);
        fprintf(stdout, "DONE\n");
        fflush(stdout);
    }
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }

    // Synchronize all streams
    try
    {
        fprintf(stdout, "Synchronizing streams........");
        fflush(stdout);

        syncStreams(transferSpecs);

        err = hipEventRecord(globalTimer->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventSynchronize(globalTimer->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        fprintf(stdout, "DONE\n");
        fflush(stdout);
    } 
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }


    // FIXME: Warn about low compute-capability here instead?

    // Print results
    fprintf(stdout, "\n");
    fprintf(stdout, "=====================================================================================\n");
    fprintf(stdout, " %2s   %-15s   %13s   %-8s   %-12s   %-10s\n",
            "ID", "Device name", "Transfer size", "Direction", "Time elapsed", "Bandwidth");
    fprintf(stdout, "-------------------------------------------------------------------------------------\n");
    fflush(stdout);

    size_t totalSize = 0;
    double aggrElapsed = .0;
    double timedElapsed = globalTimer->usecs();

    for (const TransferSpec& res : transferSpecs)
    {
        double elapsed = res.timer->usecs();
        double bandwidth = (double) res.length / elapsed;

        totalSize += res.length;
        aggrElapsed += elapsed;

        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, res.device);
        if (err != hipSuccess)
        {
            prop.name[0] = 'E';
            prop.name[1] = 'R';
            prop.name[2] = 'R';
            prop.name[3] = '!';
            prop.name[4] = '\0';
        }

        fprintf(stdout, " %2d   %-15s   %13s    %8s   %9.0f µs    %10.2f MiB/s \n",
                res.device, 
                prop.name, 
                bytesToUnit(res.length).c_str(), 
                transferDirectionToString(res.direction).c_str(),
                elapsed,
                bandwidth
               );
        fflush(stdout);
    }
    fprintf(stdout, "=====================================================================================\n");

    fprintf(stdout, "\n");
    fprintf(stdout, "Aggregated total time      : %12.0f µs\n", aggrElapsed);
    fprintf(stdout, "Aggregated total bandwidth : %12.2f MiB/s\n", (double) totalSize / aggrElapsed);
    fprintf(stdout, "Estimated elapsed time     : %12.0f µs\n", timedElapsed);
    fprintf(stdout, "Timed total bandwidth      : %12.2f MiB/s\n", (double) totalSize / timedElapsed);
    fprintf(stdout, "\n");
    fflush(stdout);
}
