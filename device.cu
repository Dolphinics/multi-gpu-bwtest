#include <hip/hip_runtime.h>
#include <exception>
#include <stdexcept>
#include <vector>
#include "device.h"

using std::runtime_error;
using std::vector;


// Cache number of devices
static int deviceCount = -1;


// Cache device properties
static vector<hipDeviceProp_t> deviceProperties;


static void loadDeviceData()
{
    hipError_t  err;

    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    deviceProperties.reserve(deviceCount);
    for (int device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, device);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        deviceProperties.push_back(prop);
    }
}


bool isDeviceValid(int device)
{
    if (deviceCount < 0)
    {
        loadDeviceData();
    }

    if (device < 0 || device >= deviceCount)
    {
        return false;
    }

    const hipDeviceProp_t& prop = deviceProperties[device];
    if (prop.computeMode == hipComputeModeProhibited)
    {
        return false;
    }

    return true;
}


int countDevices()
{
    if (deviceCount < 0)
    {
        loadDeviceData();
    }

    return deviceCount;
}


void loadDeviceProperties(int device, hipDeviceProp_t& prop)
{
    if (deviceCount < 0)
    {
        loadDeviceData();
    }

    prop = deviceProperties[device];
}
