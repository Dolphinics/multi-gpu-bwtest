#include <hip/hip_runtime.h>
#include <memory>
#include <exception>
#include <stdexcept>
#include "timer.h"


static void deleteTimer(Timer* timer)
{
    hipEventDestroy(timer->started);
    hipEventDestroy(timer->stopped);
    delete timer;
}


TimerPtr createTimer()
{
    hipError_t err;

    Timer* timer = new Timer;

    err = hipEventCreate(&timer->started);
    if (err != hipSuccess)
    {
        delete timer;
        throw std::runtime_error(hipGetErrorString(err));
    }

    err = hipEventCreate(&timer->stopped);
    if (err != hipSuccess)
    {
        hipEventDestroy(timer->started);
        delete timer;
        throw std::runtime_error(hipGetErrorString(err));
    }

    return TimerPtr(timer, &deleteTimer);
}


double Timer::usecs() const
{
    float milliseconds = .0f;

    hipError_t err = hipEventElapsedTime(&milliseconds, started, stopped);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return (double) milliseconds * 1000;
}

